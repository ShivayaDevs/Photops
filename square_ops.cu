// Corresponding header file: /include/square_ops.h
#include <hip/hip_runtime.h>

/* Write the code to square(blur) the image.
   2 cases as already specified.

   Decide the parameters for yourself and return a pointer to the new image.
   Or maybe, you can deallocate the memory of the incoming image after the operation.

   You will receive a pointer to h_inputImage so tasks like allocating memory 
   to GPU - you need to handle them yourself.  
*/


__global__ 
void square(const uchar4* d_in, uchar4* d_sq, uchar4 color, size_t numRows, size_t numCols, size_t n_numRows, size_t n_numCols)
{
 	int y = blockDim.x*blockIdx.x + threadIdx.x;	//column
	int x = blockDim.y*blockIdx.y + threadIdx.y;	//row
	int index = x*numRows + y;										//previous index of pixel
	int n_index = x*n_numRows + y;								//new index of pixel

	if(y >= n_numCols || x >= n_numRows)  				//check out of bound
	  return;

	if(y < numCols && x < numRows)								
	  d_sq[n_index] = d_in[index];
	else
	  d_sq[n_index] = color;
}

__global__ 
void square_blurr(const uchar4* d_in, uchar4* d_sq,  int blurr_amount, const float* const d_filter, const size_t filterWidth, size_t numRows, 			size_t numCols, size_t n_numRows, size_t n_numCols)
{
	//NO BLURR AMOUNT CONCEPT ADDED. WILL ADD ACCORDING TO BLURR_OP.CU FILE

	int y = blockDim.x*blockIdx.x + threadIdx.x;	//column
	int x = blockDim.y*blockIdx.y + threadIdx.y;	//row
	int index = x*numRows + y;										//previous index of pixel
	int n_index = x*n_numRows + y;								//new index of pixel

	if(y >= n_numCols || x >= n_numRows)  				//check out of bound
	  return;

	if(y < numCols && x < numRows)								
	  d_sq[n_index] = d_in[index];
	else
	{
	  int prev_x = x - (n_numRows - numRows);		//finding pixel to blurr
	  int prev_y = y - (n_numCols - numCols);
	  int prev_index = prev_x * numRows + prev_y;

	  uchar4 sum = make_uchar4(0,0,0,225);
    for(int px = 0; px < filterWidth; px++)		//calculating new pixel intensity
	  {
	    for(int py = 0; py < filterWidth; py++)
	    {
        int row = x + px - (filterWidth/2);
        int col = y + py - (filterWidth/2);
        row = min( max(0,row), static_cast<unsigned int>(numCols-1));
        col = min( max(0,col), static_cast<unsigned int>(numRows-1));
        sum.x+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].x ) );
        sum.y+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].y ) );
        sum.z+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].z ) );
	    }
  	}

  	d_sq[n_index] = sum;
	}
}

/* 
	n_numRows and n_numCols are the new row and column sizes
	d_sq represents output image intensities
*/
uchar4* square(const uchar4* const h_image, uchar4* const d_image, uchar4 color, size_t numRows, size_t numCols, 
							size_t &n_numRows, size_t &n_numCols, bool blurr, int blurr_amount, const float* const d_filter, const size_t filterWidth)
{
	size_t size, newSize;
  const dim3 blockSize(64, 64, 1);  
  const dim3 gridSize(numRows/blockSize.x+1, numCols/blockSize.y+1,1);  
  size = numRows*numCols;
  
  if(numCols > numRows)		//setting new cols and rows size
  {
    n_numRows = numCols;
    n_numCols = numCols;
  }
  else
  {
    n_numCols = numRows; 
    n_numRows = numRows;
  }
  
  newSize = n_numRows * n_numCols;
  uchar4* d_sq;
  hipMalloc(&d_sq, sizeof(uchar4)*newSize);
  if(blurr)
  square_blurr<<<gridSize, blockSize>>>(d_image, d_sq, blurr_amount, d_filter, filterWidth, numRows, numCols, n_numRows, n_numCols);
  else
  square<<<gridSize, blockSize>>>(d_image, d_sq, color, numRows, numCols, n_numRows, n_numCols);

  return d_sq;
}
