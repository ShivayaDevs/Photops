#include "hip/hip_runtime.h"
// Corresponding header file: /include/square_ops.h
#include <hip/hip_runtime.h>
#include "include/blur_ops.h"
#include <stdio.h>

/* Image squaring operations.*/

__global__ void square_kernel(uchar4 *d_in, uchar4 * d_out, size_t numRows, size_t numCols, uchar4 color){
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
 
  int width = (numCols > numRows)? numCols:numRows;
  if(x >= width || y >= width)
    return;

  if(numCols>numRows){
    int w = (numCols - numRows) / 2 ;
    if(y >= w && y < width - w)
        d_out[y*numCols + x] = d_in[(y-w)*numCols + x];
    else
      d_out[y*numCols + x] = color;
  }
  else{
    int w = (numRows - numCols) / 2 ;
    if(x >= w && x < width - w)
      d_out[y*width + x] = d_in[y*numCols + x - w];
    else
      d_out[y*width + x] = color;
  }
}

__global__
void square_blur(const uchar4* d_in, uchar4* d_blur, uchar4* d_out, size_t numRows, size_t numCols)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int width = (numRows > numCols)? numRows: numCols;

  if(x >= width || y >= width)  //check out of bounds
    return ;

  if(numCols > numRows)
  {
    int w = (numCols - numRows) / 2;
    if(y >= w && y < width - w)
      d_out[y * numCols + x] = d_in[(y-w) * numCols + x];
    else if(y < w)
      d_out[y * numCols + x] = d_blur[y * numCols + x];
    else
      d_out[y * numCols + x] = d_blur[(numRows + y - width) * numCols + x];

  }
  else
  {
    int w = (numRows - numCols) / 2;
    if(x >= w && x <= width - w )
      d_out[y * width + x] = d_in[y * numCols + (x - w)]; 
    else if(x < w)
      d_out[y * width + x] = d_blur[y * numCols + x];
    else
      d_out[y * width + x] = d_blur[y * numCols + (numCols + x - width)];
  }

}



uchar4* square_image(uchar4* const d_in, size_t &numRows, size_t &numCols, uchar4 color){

  size_t width = (numCols > numRows)? numCols : numRows;

  uchar4 *d_out;
  hipMalloc((void **) &d_out, width * width * sizeof(uchar4));

  dim3 block_size(16, 16, 1);
  dim3 grid_size(width/block_size.x + 1, width/block_size.y + 1, 1);

  square_kernel<<<grid_size, block_size>>>(d_in, d_out, numRows, numCols, color);

  numRows = numCols = width;
  uchar4 *h_out = new uchar4[width * width * sizeof(uchar4)];
  hipMemcpy(h_out, d_out, width * width * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_out);
  return h_out;   
}

void zoom(uchar4 *h_in, uchar4 *h_out, size_t numRows, size_t numCols)
{
  size_t width = (numRows > numCols)? numRows: numCols;

  double scaleWidth = (double)width/(double)numCols;
  double scaleHeight = (double)width/(double)numRows;

  for(long cy = 0; cy < width; cy++)
  {
    for(long cx = 0; cx < width; cx++)
    {
      int pixel = (cy * width) + cx;
      int y = cy/scaleHeight;
      int x = cx/scaleWidth;
      int nearest_pixel = (y * (numCols ) + x );
      
      h_out[pixel] = h_in[nearest_pixel];

    }
  }
}

void zoom_2(uchar4 *h_in, uchar4 *h_out, size_t numRows, size_t numCols, int scaleFactor)
{
  for(long cy = 0; cy < (numRows * scaleFactor); cy++)
  {
    for(long cx = 0; cx < (numCols * scaleFactor); cx++)
    {
      int pixel = (cy * numCols * scaleFactor) + cx;
      int y = cy/scaleFactor;
      int x = cx/scaleFactor;
      int nearest_pixel = (y * (numCols ) + x );
      
      h_out[pixel] = h_in[nearest_pixel];
    }
  }
}
__global__
void blur_new(const uchar4* d_in, uchar4* d_blur, uchar4* d_out, size_t numRows, size_t numCols)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int width = (numRows > numCols)? numRows: numCols;
  int w;
  if(x >= width || y>=width)
    return;
  if(numCols > numRows)
  {
    w = (numCols - numRows)/2;
    if(y >= w && y <= width - w)
      d_out[y * width + x] = d_in[(y-w) * numCols + x];
    else
      d_out[y*width + x] = d_blur[y * width + x];
  }

}

__global__
void blur_new2(uchar4 *d_in, uchar4 *d_blur, uchar4 *d_out, size_t numRows, size_t numCols)
{
  int width = (numCols > numRows)? numCols: numRows;
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if(x >= width || y >= width)
    return; 

  if(numCols > numRows)
  {
    int scaleFactor = numCols/numRows;
    int shiftFactor = ( (numCols * scaleFactor) - width ) / 2 ;
    int w = (numCols - numRows) / 2;

    if(y >= w && y < width - w)
      d_out[y * numCols + x] = d_in[(y - w) * numCols + x];
    else
      d_out[y * numCols + x] = d_blur[y * numCols * scaleFactor + (x + shiftFactor)];

  }

}

/*
output: a.jpg
kernel: blur_new
uchar4* square_blur(uchar4* d_image, size_t &numRows, size_t &numCols, int blurKernelWidth, float blurKernelSigma)
{
	size_t width = (numRows > numCols)? numRows: numCols;
  //const dim3 blockSize(16, 16, 1);  
  //const dim3 gridSize(width/blockSize.x+1, width/blockSize.y+1, 1);  
  
  // uchar4* h_blur = new uchar4[numRows * numCols * sizeof(uchar4)];

  // //calling vagisha's function
  // h_blur = blur_ops(d_image, numRows, numCols, blurKernelWidth, blurKernelSigma);
  // //h_blur = blur_ops(d_in, numRows, numCols, 9, 2.0);

  // uchar4* d_blur;
  // hipMalloc(&d_blur, sizeof(uchar4) * numRows * numCols);
  // hipMemcpy(d_blur, h_blur, sizeof(uchar4) * numRows * numCols, hipMemcpyHostToDevice);

  // uchar4* d_out;
  // hipMalloc(&d_out, sizeof(uchar4) * width * width);

  // square_blur<<<gridSize, blockSize>>>(d_image, d_blur, d_out, numRows, numCols);

  uchar4 *h_image = new uchar4[numCols * numRows * sizeof(uchar4)];
  hipMemcpy(h_image, d_image, numCols * numRows * sizeof(uchar4), hipMemcpyDeviceToHost);
  
  uchar4 *h_zoom = new uchar4[width * width * sizeof(uchar4)];
  zoom(h_image, h_zoom, numRows, numCols);
  

  uchar4 *d_in, *d_blur, *h_blur;
  hipMalloc(&d_in, sizeof(uchar4) * width * width);
  hipMemcpy(d_in, h_zoom, sizeof(uchar4) * width * width, hipMemcpyHostToDevice );

  hipMalloc(&d_blur, sizeof(uchar4) * width * width);
  h_blur = new uchar4[width * width * sizeof(uchar4)];

  h_blur = blur_ops(d_in, width, width, blurKernelWidth, blurKernelSigma);
  
  hipMemcpy(d_blur, h_blur, sizeof(uchar4) * width * width, hipMemcpyHostToDevice);
  dim3 threads(16, 16, 1);
  dim3 blocks(width/threads.x, width/threads.y, 1);

  uchar4 *d_out;
  hipMalloc(&d_out, sizeof(uchar4) * width * width);
  blur_new<<<blocks, threads>>>(d_image, d_blur, d_out, numRows, numCols);

  uchar4 *h_out = new uchar4[sizeof(uchar4) * width * width];
  hipMemcpy(h_out, d_out, width * width * sizeof(uchar4), hipMemcpyDeviceToHost);


  numRows = numCols = width;
  return h_out;
  
  //uchar4 *h_out = new uchar4[width * width * sizeof(uchar4)];
  //hipMemcpy(h_out, d_out, width * width * sizeof(uchar4), hipMemcpyDeviceToHost);
  //return h_out; 
}
*/
#include<iostream>
uchar4* square_blur(uchar4* d_image, size_t &numRows, size_t &numCols, int blurKernelWidth, float blurKernelSigma)
{
  uchar4 *h_image = new uchar4[numCols * numRows * sizeof(uchar4)];
  hipMemcpy(h_image, d_image, numRows * numCols * sizeof(uchar4), hipMemcpyDeviceToHost);

  if(numCols > numRows)
  {
    int scaleFactor = numCols/numRows + 1;
    size_t newSize = numCols * numRows * scaleFactor * scaleFactor;

    std::cout<<"Size: "<<newSize;
    std::cout<<"rows: "<<numRows * scaleFactor;
    std::cout<<"Cols: "<<numCols * scaleFactor;
    std::cout<<"factor: "<<scaleFactor;

    uchar4 *h_zoom = new uchar4[sizeof(uchar4) * newSize];

    zoom_2(h_image, h_zoom, numRows, numCols, scaleFactor);


    uchar4 *d_zoom;
    hipMalloc(&d_zoom, sizeof(uchar4) * newSize);
    hipMemcpy(d_zoom, h_zoom, sizeof(uchar4) * newSize, hipMemcpyHostToDevice);

    uchar4 *h_blur = new uchar4[sizeof(uchar4) * newSize];
    h_blur = blur_ops(d_zoom, numRows * scaleFactor, numCols * scaleFactor, blurKernelWidth, blurKernelSigma);

    uchar4 * d_blur;
    hipMalloc(&d_blur, sizeof(uchar4) * newSize);
    hipMemcpy(d_blur, h_blur, sizeof(uchar4) * newSize, hipMemcpyHostToDevice);

    size_t width = (numCols > numRows)? numCols: numRows;
    dim3 threads(16, 16, 1);
    dim3 blocks(width/threads.x, width/threads.y, 1);

    uchar4 *d_out;
    hipMalloc(&d_out, sizeof(uchar4) * width * width);
    blur_new2<<<blocks, threads>>>(d_image, d_blur, d_out, numRows, numCols);

    numCols = numRows = width;

    uchar4 *h_out = new uchar4[width * width * sizeof(uchar4)];
    hipMemcpy(h_out, d_out, sizeof(uchar4) * width * width, hipMemcpyDeviceToHost);
    return h_out;

  }
  
}
