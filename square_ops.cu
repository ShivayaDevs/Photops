#include "hip/hip_runtime.h"
// Corresponding header file: /include/square_ops.h
#include <hip/hip_runtime.h>
#include "include/blur_ops.h"

/* Write the code to square(blur) the image.
   2 cases as already specified.

   Decide the parameters for yourself and return a pointer to the new image.
   Or maybe, you can deallocate the memory of the incoming image after the operation.

   You will receive a pointer to h_inputImage so tasks like allocating memory 
   to GPU - you need to handle them yourself.  
*/


__global__ 
void square(const uchar4* d_in, uchar4* d_sq, size_t numRows, size_t numCols, size_t n_numRows, size_t n_numCols, uchar4 color)
{
 	int y = blockDim.x*blockIdx.x + threadIdx.x;	//column
	int x = blockDim.y*blockIdx.y + threadIdx.y;	//row
	int index = x*numRows + y;										//previous index of pixel
	int n_index = x*n_numRows + y;								//new index of pixel

	if(y >= n_numCols || x >= n_numRows)  				//check out of bound
	  return;

	if(y < numCols && x < numRows)								
	  d_sq[n_index] = d_in[index];
	else
	  d_sq[n_index] = color;
}

__global__ 
void square_blur(const uchar4* d_in, uchar4* d_sq, const float* const d_filter, const int filterWidth, 
                 size_t numRows, size_t numCols,  size_t n_numRows, size_t n_numCols)
{
	int y = blockDim.x*blockIdx.x + threadIdx.x;  //column
	int x = blockDim.y*blockIdx.y + threadIdx.y;	//row
	int index = x*numRows + y;										//previous index of pixel
	int n_index = x*n_numRows + y;						//new index of pixel

	if(y >= n_numCols || x >= n_numRows)  		//check out of bound
	  return;

	if(y < numCols && x < numRows)								
	  d_sq[n_index] = d_in[index];
	else
	{
	  int prev_x = x - (n_numRows - numRows);		//finding pixel to blurr
	  int prev_y = y - (n_numCols - numCols);
	  int prev_index = prev_x * numRows + prev_y;

	  uchar4 sum = make_uchar4(0,0,0,225);
    for(int px = 0; px < filterWidth; px++)		//calculating new pixel intensity
	  {
	    for(int py = 0; py < filterWidth; py++)
	    {
       	int row = x + px - (filterWidth/2);
        int col = y + py - (filterWidth/2);
        row = min( max(0,row), static_cast<unsigned int>(numCols-1));
        col = min( max(0,col), static_cast<unsigned int>(numRows-1));
        sum.x+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].x ) );
        sum.y+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].y ) );
        sum.z+= d_filter[py*filterWidth+px] * ( static_cast<float>( d_in[prev_index].z ) );
	    }
  	}

  	d_sq[n_index] = sum;
	}
}

/* 
	n_numRows and n_numCols are the new row and column sizes
	d_sq represents output image intensities
*/

uchar4* square(uchar4* const d_image, size_t numRows, size_t numCols, size_t &n_numRows, size_t &n_numCols, uchar4 color)
{
	size_t newSize;
  const dim3 blockSize(64, 64, 1);  
  const dim3 gridSize(numRows/blockSize.x+1, numCols/blockSize.y+1,1);  
   
  if(numCols > numRows)		//setting new cols and rows size
  {
    n_numRows = numCols;
    n_numCols = numCols;
  }
  else
  {
    n_numCols = numRows; 
    n_numRows = numRows;
  }
  
	newSize = n_numRows * n_numCols;
  uchar4* d_sq;
  hipMalloc(&d_sq, sizeof(uchar4)*newSize);
	square<<<gridSize, blockSize>>>(d_image, d_sq, numRows, numCols, n_numRows, n_numCols, color);

  uchar4 *h_out = new uchar4[n_numRows * n_numCols * sizeof(uchar4)];
  hipMemcpy(h_out, d_sq, n_numRows * n_numCols * sizeof(uchar4), hipMemcpyDeviceToHost);
  return h_out; 
}


uchar4* square_blur(uchar4* const d_image, size_t numRows, size_t numCols, size_t &n_numRows, size_t &n_numCols, int blurKernelWidth, float blurKernelSigma)
{
	size_t newSize;
  const dim3 blockSize(64, 64, 1);  
  const dim3 gridSize(numRows/blockSize.x+1, numCols/blockSize.y+1,1);  
  
  if(numCols > numRows)		//setting new cols and rows size
  {
    n_numRows = numCols;
    n_numCols = numCols;
  }
  else
  {
    n_numCols = numRows; 
    n_numRows = numRows;
  }
  
  newSize = n_numRows * n_numCols;
  uchar4* d_sq;
  hipMalloc(&d_sq, sizeof(uchar4)*newSize);
  
  /*setting the filter: we need to change blurkernelWidth and blurKernelSigma to change the filter */
  float* h_filter;
  size_t filterWidth;
  // setFilter(&h_filter, &filterWidth, blurKernelWidth, blurKernelSigma);

  float *d_filter; 	//creating device copy of h_filter
  hipMalloc(&d_filter, sizeof(float) * blurKernelWidth * blurKernelWidth);
  hipMemcpy(d_filter, h_filter, sizeof(float) * blurKernelWidth * blurKernelWidth, hipMemcpyHostToDevice);

  square_blur<<<gridSize, blockSize>>>(d_image, d_sq, d_filter, filterWidth, numRows, numCols, n_numRows, n_numCols);

  uchar4 *h_out = new uchar4[n_numRows * n_numCols * sizeof(uchar4)];
  hipMemcpy(h_out, d_sq, n_numRows * n_numCols * sizeof(uchar4), hipMemcpyDeviceToHost);
  return h_out; 
}
