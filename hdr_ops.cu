// Corresponding header file: /include/square_ops.h
#include <hip/hip_runtime.h>
#include <float.h>

float *d_x, *d_y, *d_logY;

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      float* const redChannel,
                      float* const greenChannel,
                      float* const blueChannel)
{
  int absolute_image_position_x = blockDim.x * blockIdx.x + threadIdx.x;
  int absolute_image_position_y = blockDim.y * blockIdx.y + threadIdx.y;

  if ( absolute_image_position_x >= numCols || absolute_image_position_y >= numRows )
  	return ;

  int thread_1D_pos = absolute_image_position_y * numCols + absolute_image_position_x;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

__global__
void rgb_to_xyY(
    float* d_r,
    float* d_g,
    float* d_b,
    float* d_x,
    float* d_y,
    float* d_log_Y,
    float  delta,
    int num_pixels_y,
    int num_pixels_x )
{
	int  ny = num_pixels_y;
  int  nx = num_pixels_x;
  int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
  int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

  if ( image_index_2d.x < nx && image_index_2d.y < ny )
  {
    float r = d_r[ image_index_1d ];
    float g = d_g[ image_index_1d ];
    float b = d_b[ image_index_1d ];

    float X = ( r * 0.4124f ) + ( g * 0.3576f ) + ( b * 0.1805f );
    float Y = ( r * 0.2126f ) + ( g * 0.7152f ) + ( b * 0.0722f );
    float Z = ( r * 0.0193f ) + ( g * 0.1192f ) + ( b * 0.9505f );

    float L = X + Y + Z;
    float x = X / L;
    float y = Y / L;

    float log_Y = log10f( delta + Y );

    d_x[ image_index_1d ]     = x;
    d_y[ image_index_1d ]     = y;
    d_log_Y[ image_index_1d ] = log_Y;
  }
}

__global__ void kernel_scan(int* d_bins, int size)
{
    int index = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(index >= size)
      return;
    int temp;
    if(index > 0)
    {
      temp = d_bins[index - 1];
    }
    else
    {
      temp = 0;
    }
    __syncthreads();
    
    d_bins[index] = temp;
    __syncthreads();
    
    int val = 0;
    for(int s=1; s<=size; s*=2)
    {
        int a = index-s;
        val = 0; 
        if(a>=0)
             val = d_bins[a];
        __syncthreads();
        
        if(a>=0)
            d_bins[index] += val;
        __syncthreads();
    }
}

__global__ void kernel_histo(const float* d_in, int* d_bins, float min,float max,int size, int numBins)
{
	int index = blockDim.x*blockIdx.x+threadIdx.x;
  if(index<size)
  {
      int a = ((d_in[index] - min)/(max-min))* numBins;
      atomicAdd(&d_bins[a], 1);
  }
}

__global__ void kernel_maxmin(float* d_in, float*d_out, int size, int maxmin)
{
    int tid = threadIdx.x;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    
    extern __shared__ float shared[];
    
    if(x>=size)
      return ;
    
    if(x<size)
    	shared[tid] = d_in[x];
    else
    {
        if(maxmin == 0)
            shared[tid] = FLT_MAX;
        else
            shared[tid] = -FLT_MAX;
    }
    __syncthreads();
    
    for(int s=1; s<blockDim.x; s++)
    {
      if(tid % (2*s) == 0)
      {
        if(s+tid < blockDim.x)
            if(maxmin == 0)
              shared[tid] = min(shared[tid], shared[tid+s]);
            else
              shared[tid] = max(shared[tid], shared[tid+s]);
        
      }
        __syncthreads();
    }
    __syncthreads();
    
    if(tid == 0)
        d_out[blockIdx.x] = shared[0];
}


__global__ void tonemap(
    float* d_x,
    float* d_y,
    float* d_log_Y,
    float* d_cdf_norm,
    float* d_r_new,
    float* d_g_new,
    float* d_b_new,
    float  min_log_Y,
    float  max_log_Y,
    float  log_Y_range,
    int    num_bins,
    int    num_pixels_y,
    int    num_pixels_x )
{
  int  ny             = num_pixels_y;
  int  nx             = num_pixels_x;
  int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
  int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

  if ( image_index_2d.x < nx && image_index_2d.y < ny )
  {
    float x         = d_x[ image_index_1d ];
    float y         = d_y[ image_index_1d ];
    float log_Y     = d_log_Y[ image_index_1d ];
    int   bin_index = min( num_bins - 1, int( (num_bins * ( log_Y - min_log_Y ) ) / log_Y_range ) );
    float Y_new     = d_cdf_norm[ bin_index ];

    float X_new = x * ( Y_new / y );
    float Z_new = ( 1 - x - y ) * ( Y_new / y );

    float r_new = ( X_new *  3.2406f ) + ( Y_new * -1.5372f ) + ( Z_new * -0.4986f );
    float g_new = ( X_new * -0.9689f ) + ( Y_new *  1.8758f ) + ( Z_new *  0.0415f );
    float b_new = ( X_new *  0.0557f ) + ( Y_new * -0.2040f ) + ( Z_new *  1.0570f );

    d_r_new[ image_index_1d ] = r_new;
    d_g_new[ image_index_1d ] = g_new;
    d_b_new[ image_index_1d ] = b_new;
  }
}

__global__
void recombineChannels(const float* const redChannel,
                       const float* const greenChannel,
                       const float* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

__global__ void normalize_cdf(
    unsigned int* d_input_cdf,
    float*        d_output_cdf,
    int           n
    )
{
  const float normalization_constant = 1.f / d_input_cdf[n - 1];

  int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( global_index_1d < n )
  {
    unsigned int input_value  = d_input_cdf[ global_index_1d ];
    float        output_value = input_value * normalization_constant;

    d_output_cdf[ global_index_1d ] = output_value;
  }
}



void findmaxmin(const float* d_in, float &ans,int size, int maxmin)
{

  int block_size = 1024;
  int curr_size = size;
  float *d_curr_in;
  hipMalloc(&d_curr_in, size * sizeof(float));
  hipMemcpy(d_curr_in, d_in, size*sizeof(float), hipMemcpyDeviceToDevice);
  float *d_out;
  while(1)
  {
    int newSize = curr_size/block_size +1;
    hipMalloc(&d_out, sizeof(float) * newSize); 
    dim3 threads(block_size);
    dim3 blocks(newSize);
    
    if(maxmin == 0)
        kernel_maxmin<<<blocks, threads, sizeof(float)*block_size>>>(d_curr_in, d_out, curr_size, 0);
    else
        kernel_maxmin<<<blocks, threads, sizeof(float)*block_size >>>(d_curr_in, d_out, curr_size, 1);
        
    hipMalloc(&d_curr_in, sizeof(float)*newSize);
    hipMemcpy(d_curr_in, d_out, sizeof(float)* newSize, hipMemcpyDeviceToDevice);
    
    if(newSize == 1)
    break;
    curr_size = newSize;
  }
  hipMemcpy(&ans, d_out, sizeof(float), hipMemcpyDeviceToHost);
}

void preprocess(uchar4* d_image, float** d_luminance, unsigned int** d_cdf, size_t &numRows, size_t &numCols, unsigned int *numberOfBins)
{
	float *d_red, *d_blue, *d_green;
	int numBins = 1024;

	dim3 blockSize(32,32,1);
	const dim3 gridSize(numRows/blockSize.x+1, numCols/blockSize.y+1,1);  

	hipMalloc(&d_red, sizeof( float) * numRows * numCols);
	hipMalloc(&d_blue, sizeof(float) * numRows * numCols);
	hipMalloc(&d_green, sizeof( float) * numRows * numCols);
	separateChannels<<<gridSize, blockSize>>>(d_image, numRows, numCols, d_red, d_blue, d_green);

	hipMalloc(&d_x, sizeof( float) * numRows * numCols);
	hipMalloc(&d_y, sizeof(float) * numRows * numCols);
	hipMalloc(&d_logY, sizeof(float) * numRows * numCols);
	rgb_to_xyY<<<gridSize, blockSize>>>(d_red, d_blue, d_green, d_x, d_y, d_logY, .0001f, numRows, numCols) ;

	*d_luminance = d_logY;

	//allocate memory for the cdf of the histogram
  *numberOfBins = numBins;
  (hipMalloc(&d_cdf, sizeof(unsigned int) * numBins));
  (hipMemset(d_cdf, 0, sizeof(unsigned int) * numBins));
}

void toneMap(const float* const d_logLuminance,
            unsigned int* const d_cdf,
            float &min_logLum,
            float &max_logLum,
            const size_t numRows,
            const size_t numCols,
            const size_t numBins)
{
	findmaxmin(d_logLuminance, min_logLum, numRows*numCols, 0);
  findmaxmin(d_logLuminance, max_logLum, numRows*numCols, 1);

  int *d_bins;
  hipMalloc(&d_bins, sizeof(int)*numBins);
  hipMemset(d_bins, 0, sizeof(int)*numBins);
  
  int size = numRows*numCols;
  dim3 threads(1024,1,1);
  dim3 blocks((size/threads.x)+1,1,1);
  
  kernel_histo<<<blocks, threads>>>(d_logLuminance, d_bins, min_logLum, max_logLum, size, numBins);
  
  dim3 scan_blocks(numBins/threads.x +1);
  kernel_scan<<<scan_blocks, threads>>>(d_bins, numBins);
  hipMemcpy(d_cdf, d_bins, sizeof(int) * numBins, hipMemcpyDeviceToDevice);
}

void postProcess(uchar4* d_out, const float* const d_logLuminance,
                unsigned int* const d_cdf,
                size_t numRows, size_t numCols,
	              float min_log_Y, float max_log_Y)
	{
	int numBins = 1024;
	const int numPixels = numRows * numCols;

  const int numThreads = 192;

  float *d_cdf_normalized;

 	hipMalloc(&d_cdf_normalized, sizeof(float) * numBins);

  normalize_cdf<<< (numBins + numThreads - 1) / numThreads,
                    numThreads>>>(d_cdf,
                                  d_cdf_normalized,
                                  numBins);
  float log_Y_range = max_log_Y - min_log_Y;

  const dim3 blockSize(32, 16, 1);
  const dim3 gridSize( (numCols + blockSize.x - 1) / blockSize.x,
                       (numRows + blockSize.y - 1) / blockSize.y );

  //next perform the actual tone-mapping
  //we map each luminance value to its new value
  //and then transform back to RGB space
  float *d_red, *d_blue, *d_green;
  hipMalloc(&d_red, sizeof( float) * numRows * numCols);
	hipMalloc(&d_blue, sizeof(float) * numRows * numCols);
	hipMalloc(&d_green, sizeof(float) * numRows * numCols);
  tonemap<<<gridSize, blockSize>>>(d_x, d_y, d_logY,
                                   d_cdf_normalized,
                                   d_red, d_green, d_blue,
                                   min_log_Y, max_log_Y,
                                   log_Y_range, numBins,
                                   numRows, numCols);

  recombineChannels<<<gridSize, blockSize>>>(d_red, d_blue, d_green, d_out, numRows, numCols);

}

uchar4* hdr_ops(uchar4* d_image, size_t numRows, size_t numCols)
{
	float *d_luminance;
  unsigned int *d_cdf;
  unsigned int numBins;
 	float min_logLum, max_logLum;
  min_logLum = 0.f;
  max_logLum = 1.f;

  preprocess(d_image, &d_luminance, &d_cdf, numRows, numCols, &numBins);

  toneMap(d_luminance, d_cdf, min_logLum, max_logLum, numRows, numCols, numBins);

  uchar4* d_out;
  hipMalloc(&d_out, sizeof(uchar4) * numRows * numCols);
  postProcess(d_out, d_luminance, d_cdf, numRows, numCols, min_logLum, max_logLum);

  return d_out;

}
