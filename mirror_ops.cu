#include "hip/hip_runtime.h"
// Corresponding header file: /include/mirror_ops.h
#include <hip/hip_runtime.h>
#include <stdio.h>    
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>   

/* Write the code to mirror the image.
   Mirror's orientation can be both horizontal as well as vertical.
   Decide the parameters for yourself and return a pointer to the new image.
   Or maybe, you can deallocate the memory of the incoming image after the operation.

   You will receive a pointer to h_in so tasks like allocating memory 
   to GPU - you need to handle them yourself.  
*/

unsigned char *d_red, *d_green, *d_blue;

__global__
void mirror(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols, bool vertical)
{

  __shared__ unsigned char sharedBlockA[4][4];   // 1. shared memory for reverse swap

  __shared__ unsigned char sharedBlockB[4][4];   // 2. shared memory for reverse swap

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if ( col >= numCols || row >= numRows )
  {
   return;
  }
  
  int tx = threadIdx.x;   // thread index X-Dir
  int ty = threadIdx.y;   // thread index Y-Dir
  
  int dX = numCols;	//  the pictures width
  int dY = numRows;	//  the pictures height

  if(vertical)
  {
  	int blockIdxA = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * dX;	 //  begin read
  	int blockIdxB = dX * dY - blockIdxA - blockDim.y * dX - blockDim.x;		 //  store data
  
  	sharedBlockA[ty][tx] = inputChannel[blockIdxA + ty * dX + tx];  // linear data fetch from global memory
		__syncthreads ();   // wait for all threads to reach this point
		sharedBlockB[ty][tx] = sharedBlockA[3-ty][3-tx]; // mirror each element in the cache
		__syncthreads();   // wait for all threads to reach this point

   	outputChannel[blockIdxB + ty * dX + tx] = sharedBlockB[ty][tx];   // linear data store in global memory
  }

  else
  {
  	int blockIdxA = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * dX;	 //  begin read
  	int blockIdxB = blockIdx.y * blockDim.y * dX + dX*blockDim.y - blockIdx.x*blockDim.x - blockDim.x*blockDim.y; //  store data
  
  	sharedBlockA[ty][tx] = inputChannel[blockIdxA + ty * dX + tx];  // linear data fetch from global memory
		__syncthreads ();   // wait for all threads to reach this point
		sharedBlockB[ty][tx] = sharedBlockA[ty][3-tx]; // mirror each element in the cache
		__syncthreads();   // wait for all threads to reach this point

   	outputChannel[blockIdxB + ty * dX + tx] = sharedBlockB[ty][tx];   // linear data store in global memory
  }

}


__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{

  
  int absolute_image_position_x = blockDim.x * blockIdx.x + threadIdx.x;
  int absolute_image_position_y = blockDim.y * blockIdx.y + threadIdx.y;

  if ( absolute_image_position_x >= numCols ||
      absolute_image_position_y >= numRows )
  {
       return;
  }
  
  int thread_1D_pos = absolute_image_position_y * numCols + absolute_image_position_x;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;

}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}




void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage)
{

  //allocate memory for the three different channels
  //original
  hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage);
  hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage);
  hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage);

}

void cleanup() {
  hipFree(d_red);
  hipFree(d_green);
  hipFree(d_blue);
}

uchar4* mirror_ops(const uchar4* const h_in, size_t numRows, size_t numCols, bool vertical)
{
	//Set reasonable block size (i.e., number of threads per block)

  const dim3 blockSize(4,4,1);
  //Calculate Grid SIze
  int a=numCols/blockSize.x, b=numRows/blockSize.y;	
  const dim3 gridSize(a+1,b+1,1);

  const size_t numPixels = numRows * numCols;

  //allocate memory on the device for both input and output
  hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels);
  hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels);
  hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4)); //make sure no memory is left laying around

  //copy input array to the GPU
  hipMemcpy(d_inputImageRGBA, h_in, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);

  //blurred
  hipMalloc(d_redBlurred, sizeof(unsigned char) * numPixels);
  hipMalloc(d_greenBlurred,  sizeof(unsigned char) * numPixels);
  hipMalloc(d_blueBlurred,   sizeof(unsigned char) * numPixels);
  hipMemset(*d_redBlurred,   0, sizeof(unsigned char) * numPixels);
  hipMemset(*d_greenBlurred, 0, sizeof(unsigned char) * numPixels);
  hipMemset(*d_blueBlurred,  0, sizeof(unsigned char) * numPixels);

  allocateMemoryAndCopyToGPU(numRows, numCols);

  //Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red,d_green, d_blue);

  hipDeviceSynchronize(); 

  //Call mirror kernel here 3 times, once for each color channel.
  mirror<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, vertical);
  mirror<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, vertical);
  mirror<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, vertical);


  hipDeviceSynchronize(); 

  //Now we recombine the results.

  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); 

  //cleanup memory
  cleanup();
  hipFree(d_redBlurred);
  hipFree(d_greenBlurred);
  hipFree(d_blueBlurred);

  hipDeviceSynchronize(); 

  //Initialize memory on host for output uchar4*
  uchar4* h_out;
  h_out = (uchar4*)malloc(sizeof(uchar4) * numPixels)

  //copy output from device to host
  hipMemcpy(h_out, d_outputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  //cleanup memory on device
  hipFree(d_inputImageRGBA);
  hipFree(d_outputImageRGBA);

  //return h_out
	return h_out;
}